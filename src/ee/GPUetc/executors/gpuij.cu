#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <string.h>
#include <sys/time.h>
#include <error.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cudaProfiler.h>
#include "gpuij.h"
#include <sys/time.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/scan.h>
#include <thrust/fill.h>
#include <inttypes.h>
#include <thrust/system/cuda/execution_policy.h>
#include "utilities.h"

namespace gpu {

GPUIJ::GPUIJ()
{
		join_result_ = NULL;
		result_size_ = 0;
		lookup_type_ = INDEX_LOOKUP_TYPE_EQ;
}

GPUIJ::GPUIJ(GTable outer_table,
				GTable inner_table,
				std::vector<ExpressionNode*> search_exp,
				ExpressionNode *end_expression,
				ExpressionNode *post_expression,
				ExpressionNode *initial_expression,
				ExpressionNode *skipNullExpr,
				ExpressionNode *prejoin_expression,
				ExpressionNode *where_expression,
				IndexLookupType lookup_type)
{
	/**** Table data *********/
	outer_table_ = outer_table;
	inner_table_ = inner_table;
	join_result_ = NULL;
	result_size_ = 0;
	lookup_type_ = lookup_type;

	/**** Expression data ****/
	search_exp_ = GExpressionVector(search_exp);

	end_expression_ = GExpression(end_expression);

	post_expression_ = GExpression(post_expression);

	initial_expression_ = GExpression(initial_expression);

	skipNullExpr_ = GExpression(skipNullExpr);

	prejoin_expression_ = GExpression(prejoin_expression);

	where_expression_ = GExpression(where_expression);
}

GPUIJ::~GPUIJ()
{
	free(join_result_);
	search_exp_.free();
	end_expression_.free();
	post_expression_.free();
	initial_expression_.free();
	skipNullExpr_.free();
	prejoin_expression_.free();
	where_expression_.free();
}

bool GPUIJ::execute(){
	gettimeofday(&all_start_, NULL);

	/******** Calculate size of blocks, grids, and GPU buffers *********/
	uint gpu_size = 0, part_size = 0;
	ulong jr_size;

	RESULT *jresult_dev, *write_dev;
	jresult_dev = write_dev = NULL;
	ulong *index_psum, *exp_psum;
	ResBound *res_bound;
	bool *prejoin_res_dev;

	part_size = getPartitionSize();

//	int block_x, grid_x;
//
//	block_x = (part_size < BLOCK_SIZE_X) ? part_size : BLOCK_SIZE_X;
//	grid_x = (part_size - 1)/block_x + 1;
	gpu_size = DEFAULT_PART_SIZE_ + 1;

	/******** Allocate GPU buffer for table data and counting data *****/
	checkCudaErrors(hipMalloc(&prejoin_res_dev, part_size * sizeof(bool)));
	checkCudaErrors(hipMalloc(&index_psum, gpu_size * sizeof(ulong)));

	checkCudaErrors(hipMalloc(&res_bound, gpu_size * sizeof(ResBound)));

	struct timeval pre_start, pre_end, istart, iend, estart, eend, pestart, peend, wstart, wend, end_join, balance_start, balance_end;

	/*** Loop over outer tuples and inner tuples to copy table data to GPU buffer **/
	for (int outer_idx = 0; outer_idx < outer_table_.getBlockNum(); outer_idx++) {
		//Size of outer small table
		outer_table_.moveToBlock(outer_idx);
		gpu_size = outer_table_.getCurrentRowNum() + 1;

		/* Evaluate prejoin predicate */
		gettimeofday(&pre_start, NULL);
		PrejoinFilter(prejoin_res_dev);
		gettimeofday(&pre_end, NULL);
		prejoin_.push_back(timeDiff(pre_start, pre_end));

		joins_only_.push_back(timeDiff(pre_start, pre_end));

		for (int inner_idx = 0; inner_idx < inner_table_.getBlockNum(); inner_idx++) {
			/* Binary search for index */
			inner_table_.moveToBlock(inner_idx);
			gettimeofday(&istart, NULL);

			IndexFilter(index_psum, res_bound, prejoin_res_dev);

			gettimeofday(&iend, NULL);
			index_.push_back(timeDiff(istart, iend));

			RESULT *tmp_result;
			ulong tmp_size = 0;

			gettimeofday(&balance_start, NULL);
			Rebalance(index_psum, res_bound, &tmp_result, gpu_size, &tmp_size);
			gettimeofday(&balance_end, NULL);

			rebalance_.push_back(timeDiff(balance_start, balance_end));

			if (tmp_size == 0) {
				gettimeofday(&end_join, NULL);
				joins_only_.push_back(timeDiff(istart, end_join));
				continue;
			}
			checkCudaErrors(hipMalloc(&jresult_dev, tmp_size * sizeof(RESULT)));
			checkCudaErrors(hipMalloc(&exp_psum, (tmp_size + 1) * sizeof(ulong)));

			gettimeofday(&estart, NULL);
			ExpressionFilter(tmp_result, jresult_dev, exp_psum, tmp_size);
			gettimeofday(&eend, NULL);

			expression_.push_back(timeDiff(estart, eend));

			gettimeofday(&pestart, NULL);
			GUtilities::ExclusiveScan(exp_psum, tmp_size + 1, &jr_size);
			gettimeofday(&peend, NULL);

			epsum_.push_back(timeDiff(pestart, peend));

			checkCudaErrors(hipFree(tmp_result));

			if (jr_size == 0) {
				continue;
			}
			checkCudaErrors(hipMalloc(&write_dev, jr_size * sizeof(RESULT)));

			gettimeofday(&wstart, NULL);
			GUtilities::RemoveEmptyResult(write_dev, jresult_dev, exp_psum, tmp_size);
			gettimeofday(&wend, NULL);
			wtime_.push_back(timeDiff(wstart, wend));

			join_result_ = (RESULT *)realloc(join_result_, (result_size_ + jr_size) * sizeof(RESULT));

			gettimeofday(&end_join, NULL);
			checkCudaErrors(hipMemcpy(join_result_ + result_size_, write_dev, jr_size * sizeof(RESULT), hipMemcpyDeviceToHost));

			result_size_ += jr_size;
			jr_size = 0;

			joins_only_.push_back(timeDiff(istart, end_join));
		}
	}

	checkCudaErrors(hipDeviceSynchronize());
	/******** Free GPU memory, unload module, end session **************/

	checkCudaErrors(hipFree(res_bound));
	checkCudaErrors(hipFree(prejoin_res_dev));
	gettimeofday(&all_end_, NULL);

	//exit(0);
	return true;
}

bool GPUIJ::execute2()
{
	hipStream_t s;



}

void GPUIJ::getResult(RESULT *output) const
{
	memcpy(output, join_result_, sizeof(RESULT) * result_size_);
}

int GPUIJ::getResultSize() const
{
	return result_size_;
}

uint GPUIJ::getPartitionSize() const
{
	int part_size = DEFAULT_PART_SIZE_;
	int outer_size = outer_table_.getCurrentRowNum();
	int inner_size = inner_table_.getCurrentRowNum();
	int bigger_tuple_size = (outer_size > inner_size) ? outer_size : inner_size;

	if (bigger_tuple_size < part_size) {
		return bigger_tuple_size;
	}

	for (uint i = 32768; i <= DEFAULT_PART_SIZE_; i = i * 2) {
		if (bigger_tuple_size < i) {
			part_size = i;
			break;
		}
	}

	printf("getPartitionSize: PART SIZE = %d\n", part_size);
	return part_size;
}


void GPUIJ::debug(void)
{
	printf("******** Debugging information *********** \n");
	printf("EXPRESSIONS:\n");

	printf("End Expression: ");
	end_expression_.debug();

	printf("Post Expression: ");
	post_expression_.debug();

	printf("Initial Expression: ");
	initial_expression_.debug();

	printf("Skip Null Expression: ");
	skipNullExpr_.debug();

	printf("Where Expression: ");
	where_expression_.debug();

	printf("\nTABLES:\n");
	printf("Outer table:");
	outer_table_.debug();

	printf("Inner table:");
	inner_table_.debug();
}


void GPUIJ::profiling()
{
	unsigned long allocation_time = 0, prejoin_time = 0, index_time = 0, expression_time = 0, ipsum_time = 0, epsum_time = 0, wtime_time = 0, joins_only_time = 0, all_time = 0;

	for (int i = 0; i < prejoin_.size(); i++) {
		prejoin_time += prejoin_[i];
	}

	for (int i = 0; i < index_.size(); i++) {
		index_time += index_[i];
	}

	for (int i = 0; i < expression_.size(); i++) {
		expression_time += expression_[i];
	}

	for (int i = 0; i < ipsum_.size(); i++) {
		ipsum_time += ipsum_[i];
	}

	for (int i = 0; i < epsum_.size(); i++) {
		epsum_time += epsum_[i];
	}

	for (int i = 0; i < wtime_.size(); i++) {
		wtime_time += wtime_[i];
	}

#if (defined(DECOMPOSED1_) || defined(DECOMPOSED2_))
	unsigned long rebalance_cost = 0;
	for (int i = 0; i < rebalance_.size(); i++) {
		rebalance_cost += rebalance_[i];
	}
#endif


	for (int i = 0; i < joins_only_.size(); i++) {
		joins_only_time += joins_only_[i];
	}

	all_time = (all_end_.tv_sec - all_start_.tv_sec) * 1000000 + (all_end_.tv_usec - all_start_.tv_usec);

	allocation_time = all_time - joins_only_time;
	printf("**********************************\n"
			"Allocation & data movement time: %lu\n"
			"Prejoin filter Time: %lu\n"
			"Index Search Time: %lu\n"
			"Rebalance Cost: %lu\n"
			"Expression filter Time: %lu\n"
			"Expression Prefix Sum Time: %lu\n"
			"Write back time Time: %lu\n"
			"Joins Only Time: %lu\n"
			"Total join time: %lu\n"
			"*******************************\n",
			allocation_time, prejoin_time, index_time,
			rebalance_cost, expression_time, epsum_time, wtime_time, joins_only_time, all_time);

}

unsigned long GPUIJ::timeDiff(struct timeval start, struct timeval end)
{
	return GUtilities::timeDiff(start, end);
}

extern "C" __global__ void PrejoinFilterDev(GTable outer, int outer_rows, GExpression prejoin, bool *result,int64_t *val_stack, ValueType *type_stack)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int offset = blockDim.x * gridDim.x;

	GTuple outer_tuple;

	for (int i = index; i < outer_rows; i+= offset) {
		GNValue res = GNValue::getTrue();
		outer_tuple = outer.getGTuple(i);

		res = (prejoin.getSize() > 1) ? prejoin.evaluate(&outer_tuple, NULL, val_stack + index, type_stack + index, offset) : res;
		result[i] = res.isTrue();
	}
}

void GPUIJ::PrejoinFilter(bool *result)
{
	int outer_rows = outer_table_.getCurrentRowNum();
	int block_x, grid_x;

	block_x = (outer_rows < BLOCK_SIZE_X) ? outer_rows : BLOCK_SIZE_X;
	grid_x = (outer_rows - 1)/block_x + 1;

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);

	PrejoinFilterDev<<<grid_size, block_size>>>(outer_table_, outer_rows, prejoin_expression_, result,val_stack, type_stack);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

void GPUIJ::PrejoinFilter(bool *result, hipStream_t stream)
{
	int outer_rows = outer_table_.getCurrentRowNum();
	int block_x, grid_x;

	block_x = (outer_rows < BLOCK_SIZE_X) ? outer_rows : BLOCK_SIZE_X;
	grid_x = (outer_rows - 1)/block_x + 1;

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);

	PrejoinFilterDev<<<grid_size, block_size, 0, stream>>>(outer_table_, outer_rows, prejoin_expression_, result, val_stack, type_stack);
	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

extern "C" __global__ void decompose(ResBound *in, RESULT *out, ulong *in_location, ulong *local_offset, int size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = index; i < size; i += blockDim.x * gridDim.x) {
		out[i].lkey = in[in_location[i]].outer;
		out[i].rkey = in[in_location[i]].left + local_offset[i];
	}
}

void GPUIJ::Decompose(ResBound *in, RESULT *out, ulong *in_location, ulong *local_offset, int size)
{
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size - 1)/block_x + 1;

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);

	decompose<<<grid_size, block_x>>>(in, out, in_location, local_offset, size);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

void GPUIJ::Decompose(ResBound *in, RESULT *out, ulong *in_location, ulong *local_offset, int size, hipStream_t stream)
{
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size - 1)/block_x + 1;

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);

	decompose<<<grid_size, block_x, 0, stream>>>(in, out, in_location, local_offset, size);
	checkCudaErrors(hipGetLastError());
}


extern "C" __global__ void IndexFilterLowerBound(GTable search_table, GTreeIndex inner_idx,
													int search_rows, int inner_rows,
													ulong *index_psum, ResBound *res_bound,
													IndexLookupType lookup_type,
													bool *prejoin_res_dev,
													int64_t *val_stack,
													ValueType *type_stack
										  	  	  )

{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	GTreeIndexKey outer_key;

	for (int i = index; i < search_rows; i += offset) {
		res_bound[i].left = -1;
		res_bound[i].outer = -1;

		if (prejoin_res_dev[i]) {
			res_bound[i].outer = i;

			GTuple tuple = search_table.getGTuple(i);
			outer_key.createKey(tuple);

			switch (lookup_type) {
			case INDEX_LOOKUP_TYPE_EQ:
			case INDEX_LOOKUP_TYPE_GT:
			case INDEX_LOOKUP_TYPE_GTE:
			case INDEX_LOOKUP_TYPE_LT: {
				res_bound[i].left = inner_idx.lowerBound(outer_key, 0, inner_rows - 1);
				break;
			}
			case INDEX_LOOKUP_TYPE_LTE: {
				res_bound[i].left = 0;
				break;
			}
			default:
				break;
			}
		}
	}
}

extern "C" __global__ void IndexFilterUpperBound(GTable search_table, GTreeIndex inner_idx,
													int search_rows, int inner_rows,
													ulong *index_psum, ResBound *res_bound,
													IndexLookupType lookup_type,
													bool *prejoin_res_dev,
													int64_t *val_stack,
													ValueType *type_stack
										  	  	  )

{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	GTreeIndexKey outer_key;

	for (int i = index; i < search_rows; i += offset) {
		index_psum[i] = 0;
		res_bound[i].right = -1;

		if (prejoin_res_dev[i]) {
			GTuple tuple = search_table.getGTuple(i);
			outer_key.createKey(tuple);

			switch (lookup_type) {
			case INDEX_LOOKUP_TYPE_EQ:
			case INDEX_LOOKUP_TYPE_LTE: {
				res_bound[i].right = inner_idx.upperBound(outer_key, 0, inner_rows - 1);
				break;
			}
			case INDEX_LOOKUP_TYPE_GT:
			case INDEX_LOOKUP_TYPE_GTE: {
				res_bound[i].right = inner_rows;
				break;
			}
			case INDEX_LOOKUP_TYPE_LT: {
				res_bound[i].right = res_bound[i].left - 1;
				res_bound[i].left = 0;
				break;
			}
			default:
				break;
			}
		}

		index_psum[i] = (res_bound[i].right >= 0 && res_bound[i].left >= 0) ? (res_bound[i].right - res_bound[i].left + 1) : 0;
	}

	if (index == 0)
		index_psum[search_rows] = 0;
}


extern "C" __global__ void constructSearchTable(GTable outer_table, GTable search_table,
												int outer_rows,
												GExpressionVector search_exp,
												int64_t *val_stack, ValueType *type_stack)
{
	GNValue tmp;
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	GTuple tuple;

	for (int i = index; i < outer_rows; i += stride) {
		tuple = search_table.getGTuple(i);
		for (int j = 0; j < search_exp.size(); j++) {
			tmp = search_exp.at(j).evaluate(&tuple, NULL, val_stack, type_stack, stride);
			tuple.setGNValue(tmp, j);
		}
	}
}

void GPUIJ::IndexFilter(ulong *index_psum, ResBound *res_bound, bool *prejoin_res_dev)
{
	int outer_rows = outer_table_.getCurrentRowNum(), inner_rows = inner_table_.getCurrentRowNum();
	int block_x, grid_x;

	block_x = (outer_rows < BLOCK_SIZE_X) ? outer_rows : BLOCK_SIZE_X;
	grid_x = (outer_rows - 1)/block_x + 1;

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);
	GTable search_table(NULL, search_exp_.size());
	GTreeIndex *inner_idx = static_cast<GTreeIndex*>(inner_table_.getCurrentIndex());

	constructSearchTable<<<grid_size, block_size>>>(outer_table_, search_table, outer_rows, search_exp_, val_stack, type_stack);

	IndexFilterLowerBound<<<grid_size, block_size>>>(search_table, *inner_idx, outer_rows, inner_rows, index_psum, res_bound, lookup_type_, prejoin_res_dev, val_stack, type_stack);

	IndexFilterUpperBound<<<grid_size, block_size>>>(outer_table_, *inner_idx, outer_rows, inner_rows, index_psum, res_bound, lookup_type_, prejoin_res_dev, val_stack, type_stack);

	checkCudaErrors(hipDeviceSynchronize());

	search_table.removeTable();

	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));

}

void GPUIJ::IndexFilter(ulong *index_psum, ResBound *res_bound, bool *prejoin_res_dev, hipStream_t stream)
{
	int outer_rows = outer_table_.getCurrentRowNum(), inner_rows = inner_table_.getCurrentRowNum();
	int block_x, grid_x;

	block_x = (outer_rows < BLOCK_SIZE_X) ? outer_rows : BLOCK_SIZE_X;
	grid_x = (outer_rows - 1) / block_x + 1;

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);
	GTable search_table(NULL, search_exp_.size());
	GTreeIndex *inner_idx = static_cast<GTreeIndex*>(inner_table_.getCurrentIndex());

	constructSearchTable<<<grid_size, block_size, 0, stream>>>(outer_table_, search_table, outer_rows, search_exp_, val_stack, type_stack);

	IndexFilterLowerBound<<<grid_size, block_size, 0, stream>>>(search_table, *inner_idx, outer_rows, inner_rows, index_psum, res_bound, lookup_type_, prejoin_res_dev, val_stack, type_stack);

	IndexFilterUpperBound<<<grid_size, block_size, 0, stream>>>(search_table, *inner_idx, outer_rows, inner_rows, index_psum, res_bound, lookup_type_, prejoin_res_dev, val_stack, type_stack);

	//checkCudaErrors(hipStreamSynchronize(stream));

	search_table.removeTable();
	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}


extern "C" __global__ void ExpressionFilterDev2(GTable outer, GTable inner,
												RESULT *in_bound, RESULT *out_bound,
												ulong *mark_location, int size,
												GExpression end_exp, GExpression post_exp, GExpression where_exp,
												int64_t *val_stack, ValueType *type_stack
												)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	GNValue res;
	GTuple outer_tuple, inner_tuple;

	for (int i = index; i < size; i += offset) {
		res = GNValue::getTrue();
		outer_tuple = outer.getGTuple(in_bound[i].lkey);
		inner_tuple = inner.getGTuple(in_bound[i].rkey);
		res = (end_exp.getSize() > 0) ? end_exp.evaluate(&outer_tuple, &inner_tuple, val_stack + index, type_stack + index, offset) : res;
		res = (post_exp.getSize() > 0 && res.isTrue()) ? post_exp.evaluate(&outer_tuple, &inner_tuple, val_stack + index, type_stack + index, offset) : res;
		res = (where_exp.getSize() > 0 && res.isTrue()) ? where_exp.evaluate(&outer_tuple, &inner_tuple, val_stack + index, type_stack + index, offset) : res;

		out_bound[i].lkey = (res.isTrue()) ? in_bound[i].lkey : (-1);
		out_bound[i].rkey = (res.isTrue()) ? in_bound[i].rkey : (-1);
		mark_location[i] = (res.isTrue()) ? 1 : 0;
	}

	if (index == 0) {
		mark_location[size] = 0;
	}
}

void GPUIJ::ExpressionFilter(RESULT *in_bound, RESULT *out_bound, ulong *mark_location, int size)
{
	int partition_size = DEFAULT_PART_SIZE_;

	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size <= partition_size) ? (size - 1)/block_x + 1 : (partition_size - 1)/block_x + 1;

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);

	ExpressionFilterDev2<<<grid_size, block_size>>>(outer_table_, inner_table_,
													in_bound, out_bound,
													mark_location, size,
													end_expression_, post_expression_, where_expression_,
													val_stack, type_stack
												);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

void GPUIJ::ExpressionFilter(RESULT *in_bound, RESULT *out_bound, ulong *mark_location, int size, hipStream_t stream)
{
	int partition_size = DEFAULT_PART_SIZE_;

	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size <= partition_size) ? (size - 1)/block_x + 1 : (partition_size - 1)/block_x + 1;

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);

	ExpressionFilterDev2<<<grid_size, block_size, 0, stream>>>(outer_table_, inner_table_,
																in_bound, out_bound,
																mark_location, size,
																end_expression_, post_expression_, where_expression_,
																val_stack, type_stack
																);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

extern "C" __global__ void ExpressionFilterDev(GTable outer, GTable inner,
												int outer_rows,
												RESULT *result, ulong *index_psum,
												ulong *exp_psum, uint result_size,
												GExpression end_dev, GExpression post_dev, GExpression where_dev,
												ResBound *res_bound, bool *prejoin_res_dev,
												int64_t *val_stack,
												ValueType *type_stack
												)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	GTuple outer_tuple, inner_tuple;

	for (int i = index; i < outer_rows; i += offset) {
		exp_psum[i] = 0;
		ulong writeloc = index_psum[index];
		int count = 0;
		int res_left = -1, res_right = -1;
		GNValue res = GNValue::getTrue();

		res_left = res_bound[i].left;
		res_right = res_bound[i].right;

		while (res_left >= 0 && res_left <= res_right && writeloc < result_size) {
			outer_tuple = outer.getGTuple(res_left);
			inner_tuple = inner.getGTuple(res_right);

			res = (end_dev.getSize() > 0) ? end_dev.evaluate(&outer_tuple, &inner_tuple, val_stack + index, type_stack + index, offset) : res;
			res = (post_dev.getSize() > 0 && res.isTrue()) ? end_dev.evaluate(&outer_tuple, &inner_tuple, val_stack + index, type_stack + index, offset) : res;
			res = (where_dev.getSize() > 0 && res.isTrue()) ? where_dev.evaluate(&outer_tuple, &inner_tuple, val_stack + index, type_stack + index, offset) : res;

			result[writeloc].lkey = (res.isTrue()) ? i : (-1);
			result[writeloc].rkey = (res.isTrue()) ? res_left : (-1);
			count += (res.isTrue()) ? 1 : 0;
			writeloc++;
			res_left++;
		}
		exp_psum[i] = count;
	}

	if (index == 0) {
		exp_psum[outer_rows] = 0;
	}
}

void GPUIJ::ExpressionFilter(ulong *index_psum, ulong *exp_psum, RESULT *result, int result_size, ResBound *res_bound, bool *prejoin_res_dev)
{
	int outer_rows = outer_table_.getCurrentRowNum();
	int partition_size = DEFAULT_PART_SIZE_;
	int block_x, grid_x;

	block_x = (outer_rows < BLOCK_SIZE_X) ? outer_rows : BLOCK_SIZE_X;
	grid_x = (outer_rows < partition_size) ? (outer_rows - 1)/block_x + 1 : (partition_size - 1)/block_x + 1;

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);

	ExpressionFilterDev<<<grid_size, block_size>>>(outer_table_, inner_table_,
													outer_rows,
													result, index_psum,
													exp_psum,
													result_size,
													end_expression_, post_expression_, where_expression_,
													res_bound, prejoin_res_dev,
													val_stack, type_stack
													);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));

}

void GPUIJ::ExpressionFilter(ulong *index_psum, ulong *exp_psum, RESULT *result, int result_size, ResBound *res_bound, bool *prejoin_res_dev, hipStream_t stream)
{
	int outer_rows = outer_table_.getCurrentRowNum();
	int partition_size = DEFAULT_PART_SIZE_;
	int block_x, grid_x;

	block_x = (outer_rows < BLOCK_SIZE_X) ? outer_rows : BLOCK_SIZE_X;
	grid_x = (outer_rows < partition_size) ? (outer_rows - 1)/block_x + 1 : (partition_size - 1)/block_x + 1;


	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 grid_size(grid_x, 1, 1);
	dim3 block_size(block_x, 1, 1);

	ExpressionFilterDev<<<grid_size, block_size, 0, stream>>>(outer_table_, inner_table_,
																outer_rows,
																result, index_psum,
																exp_psum, result_size,
																end_expression_, post_expression_, where_expression_,
																res_bound, prejoin_res_dev,
																val_stack, type_stack
																);

	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipStreamSynchronize(stream));

	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

void GPUIJ::Rebalance(ulong *in, ResBound *in_bound, RESULT **out_bound, int in_size, ulong *out_size, hipStream_t stream)
{
	GUtilities::ExclusiveScan(in, in_size, out_size, stream);

	if (*out_size == 0) {
		return;
	}

	ulong *location;

	checkCudaErrors(hipMalloc(&location, sizeof(ulong) * (*out_size)));

	checkCudaErrors(hipMemsetAsync(location, 0, sizeof(ulong) * (*out_size), stream));

	GUtilities::MarkLocation(location, in, in_size, stream);

	GUtilities::InclusiveScan(location, *out_size, stream);

	ulong *local_offset;

	checkCudaErrors(hipMalloc(&local_offset, *out_size * sizeof(ulong)));
	checkCudaErrors(hipMalloc(out_bound, *out_size * sizeof(RESULT)));

	GUtilities::ComputeOffset(in, location, local_offset, *out_size, stream);

	Decompose(in_bound, *out_bound, location, local_offset, *out_size, stream);

	checkCudaErrors(hipFree(local_offset));
	checkCudaErrors(hipFree(location));
}

void GPUIJ::Rebalance(ulong *in, ResBound *in_bound, RESULT **out_bound, int in_size, ulong *out_size)
{
	GUtilities::ExclusiveScan(in, in_size, out_size);

	if (*out_size == 0) {
		return;
	}

	ulong *location;

	checkCudaErrors(hipMalloc(&location, sizeof(ulong) * (*out_size)));

	checkCudaErrors(hipMemset(location, 0, sizeof(ulong) * (*out_size)));

	checkCudaErrors(hipDeviceSynchronize());

	GUtilities::MarkLocation(location, in, in_size);

	GUtilities::InclusiveScan(location, *out_size);

	ulong *local_offset;

	checkCudaErrors(hipMalloc(&local_offset, *out_size * sizeof(ulong)));
	checkCudaErrors(hipMalloc(out_bound, *out_size * sizeof(RESULT)));

	GUtilities::ComputeOffset(in, location, local_offset, *out_size);

	Decompose(in_bound, *out_bound, location, local_offset, *out_size);

	checkCudaErrors(hipFree(local_offset));
	checkCudaErrors(hipFree(location));
}
}
