#include "hip/hip_runtime.h"
#include "gpuhj.h"
#include "types.h"
#include "gtable.h"

#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <string.h>
#include <sys/time.h>
#include <error.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <cudaProfiler.h>
#include <math.h>


#include <inttypes.h>

#include "utilities.h"

namespace gpu {



const uint64_t GPUHJ::MAX_BUCKETS[] = {
	        3,				//0
	        7,				//1
	        13,				//2
	        31,				//3
	        61,				//4
	        127,			//5
	        251,			//6
	        509,			//7
	        1021,			//8
	        2039,			//9
	        4093,			//10
	        8191,			//11
	        16381,			//12
	        32749,			//13
	        65521,			//14
	        131071,			//15
	        262139,			//16
	        524287,			//17
	        1048573,		//18
	        2097143,		//19
	        4194301,		//20
	        8388593,		//21
	        16777213,
	        33554393,
	        67108859,
	        134217689,
	        268435399,
	        536870909,
	        1073741789,
	        2147483647,
	        4294967291,
	        8589934583
	};

GPUHJ::GPUHJ()
{
		join_result_ = NULL;
		result_size_ = 0;
		maxNumberOfBuckets_ = 0;
		total_ = 0;

		m_sizeIndex_ = 0;
		lookup_type_ = INDEX_LOOKUP_TYPE_EQ;
}

GPUHJ::GPUHJ(GTable outer_table,
				GTable inner_table,
				std::vector<ExpressionNode*> search_exp,
				ExpressionNode *end_expression,
				ExpressionNode *post_expression,
				ExpressionNode *initial_expression,
				ExpressionNode *skipNullExpr,
				ExpressionNode *prejoin_expression,
				ExpressionNode *where_expression,
				IndexLookupType lookup_type,
				int mSizeIndex)
{
	/**** Table data *********/
	outer_table_ = outer_table;
	inner_table_ = inner_table;
	join_result_ = NULL;
	result_size_ = 0;
	lookup_type_ = lookup_type;
	m_sizeIndex_ = mSizeIndex;
	total_ = 0;

	//Fix the size of bucket at 16
	maxNumberOfBuckets_ = MAX_BUCKETS[m_sizeIndex_];

	printf("New M_SIZE_INDEX = %d\n", m_sizeIndex_);

	/**** Expression data ****/

	search_exp_ = GExpressionVector(search_exp);
	end_expression_ = GExpression(end_expression);
	post_expression_ = GExpression(post_expression);
	initial_expression_ = GExpression(initial_expression);
	skipNullExpr_ = GExpression(skipNullExpr);
	prejoin_expression_ = GExpression(prejoin_expression);
	where_expression_ = GExpression(where_expression);
}

GPUHJ::~GPUHJ()
{
	free(join_result_);
	search_exp_.free();
	end_expression_.free();
	post_expression_.free();
	initial_expression_.free();
	skipNullExpr_.free();
	prejoin_expression_.free();
	where_expression_.free();
}


void GPUHJ::getResult(RESULT *output) const
{
	memcpy(output, join_result_, sizeof(RESULT) * result_size_);
}

int GPUHJ::getResultSize() const
{
	return result_size_;
}


void GPUHJ::debug(void)
{

	printf("******** Debugging information *********** \n");
	printf("EXPRESSIONS:\n");

	printf("End Expression: ");
	end_expression_.debug();

	printf("Post Expression: ");
	post_expression_.debug();

	printf("Initial Expression: ");
	initial_expression_.debug();

	printf("Skip Null Expression: ");
	skipNullExpr_.debug();

	printf("Where Expression: ");
	where_expression_.debug();

	printf("\nTABLES:\n");
	printf("Outer table:");
	outer_table_.debug();

	printf("Inner table:");
	inner_table_.debug();
}




uint GPUHJ::getPartitionSize() const
{
//	return PART_SIZE_;
	uint part_size = DEFAULT_PART_SIZE_;
//	uint outer_size = outer_rows_;
//	uint inner_size = inner_rows_;
//	uint bigger_tuple_size = (outer_size > inner_size) ? outer_size : inner_size;
//
//	if (bigger_tuple_size < part_size) {
//		return bigger_tuple_size;
//	}
//
//	for (uint i = 32768; i <= DEFAULT_PART_SIZE_; i = i * 2) {
//		if (bigger_tuple_size < i) {
//			part_size = i;
//			break;
//		}
//	}
//
//	printf("getPartitionSize: PART SIZE = %d\n", part_size);
	return part_size;
}


bool GPUHJ::join()
{


	checkCudaErrors(hipProfilerStart());
	ulong *index_count, jr_size;
	RESULT *jresult_dev;
	struct timeval start_all, end_all;

	int partition_size;

	struct timeval index_count_start, index_count_end, prefix_start, prefix_end, join_start, join_end, rebalance_start, rebalance_end, remove_start, remove_end;

	gettimeofday(&start_all, NULL);

	/******* Hash the outer table *******/

	partition_size = getPartitionSize();
	checkCudaErrors(hipMalloc(&index_count, sizeof(ulong) * (partition_size + 1)));

	ResBound *in_bound;

	checkCudaErrors(hipMalloc(&in_bound, sizeof(ResBound) * partition_size));

	printf("Start Joining\n");

	for (int outer_idx = 0; outer_idx < outer_table_.getBlockNum(); outer_idx++) {
		outer_table_.moveToBlock(outer_idx);

		for (int inner_idx = 0; inner_idx < inner_table_.getBlockNum(); inner_idx++) {
			inner_table_.moveToBlock(inner_idx);

			gettimeofday(&index_count_start, NULL);
			IndexCount(index_count, in_bound);
			gettimeofday(&index_count_end, NULL);

			index_hcount_.push_back(GUtilities::timeDiff(index_count_start, index_count_end));

			RESULT *tmp_bound, *out_bound;
			ulong out_size;
			ulong *exp_psum;

			gettimeofday(&rebalance_start, NULL);
			Rebalance(index_count, in_bound, &tmp_bound, outer_table_.getCurrentRowNum() + 1, &out_size);
			gettimeofday(&rebalance_end, NULL);
			rebalance_cost_.push_back(GUtilities::timeDiff(rebalance_start, rebalance_end));

			if (out_size == 0) {
				continue;
			}

			printf("out_size = %lu\n", out_size);
			checkCudaErrors(hipMalloc(&exp_psum, (out_size + 1) * sizeof(ulong)));
			checkCudaErrors(hipMalloc(&out_bound, out_size * sizeof(RESULT)));

			gettimeofday(&join_start, NULL);
			HashJoinLegacy(tmp_bound, out_bound, exp_psum, out_size);
			gettimeofday(&join_end, NULL);
			join_time_.push_back(GUtilities::timeDiff(join_start, join_end));

			gettimeofday(&prefix_start, NULL);
			GUtilities::ExclusiveScan(exp_psum, out_size + 1, &jr_size);
			gettimeofday(&prefix_end, NULL);

			prefix_sum_.push_back(GUtilities::timeDiff(prefix_start, prefix_end));

			checkCudaErrors(hipFree(tmp_bound));

			if (jr_size == 0) {
				checkCudaErrors(hipFree(exp_psum));
				checkCudaErrors(hipFree(out_bound));
				continue;
			}

			checkCudaErrors(hipMalloc(&jresult_dev, jr_size * sizeof(RESULT)));

			gettimeofday(&remove_start, NULL);
			GUtilities::RemoveEmptyResult(jresult_dev, out_bound, exp_psum, out_size);
			gettimeofday(&remove_end, NULL);
			remove_empty_.push_back(GUtilities::timeDiff(remove_start, remove_end));

			join_result_ = (RESULT *)realloc(join_result_, (result_size_ + jr_size) * sizeof(RESULT));

			checkCudaErrors(hipMemcpy(join_result_ + result_size_, jresult_dev, jr_size * sizeof(RESULT), hipMemcpyDeviceToHost));
#ifdef DECOMPOSED1_
			checkCudaErrors(hipFree(exp_psum));
			checkCudaErrors(hipFree(out_bound));
#endif
			checkCudaErrors(hipFree(jresult_dev));
			result_size_ += jr_size;
			jr_size = 0;
		}
	}

	gettimeofday(&end_all, NULL);

	checkCudaErrors(hipFree(index_count));

	checkCudaErrors(hipProfilerStop());

	total_ = GUtilities::timeDiff(start_all, end_all);
	return true;
}

void GPUHJ::profiling()
{
	unsigned long index_count_final, prefix_sum_final, join_final;
	unsigned long rebalance_final, remove_empty_total;

	index_count_final = 0;
	for (int i = 0; i < index_hcount_.size(); i++) {
		index_count_final += index_hcount_[i];
	}

	prefix_sum_final = 0;
	for (int i = 0; i < prefix_sum_.size(); i++) {
		prefix_sum_final += prefix_sum_[i];
	}

	rebalance_final = 0;
	for (int i = 0; i < rebalance_cost_.size(); i++) {
		rebalance_final += rebalance_cost_[i];
	}

	remove_empty_total = 0;
	for (int i = 0; i < remove_empty_.size(); i++) {
		remove_empty_total += remove_empty_[i];
	}

	join_final = 0;
	for (int i = 0; i < join_time_.size(); i++) {
		join_final += join_time_[i];
	}

	ulong join_total, data_copy;

	join_total = index_count_final + prefix_sum_final + join_final + remove_empty_total;

	data_copy = total_ - join_total;

	printf("\n*** Execution time *****************************\n"
			"index Count: %lu\n"
			"prefix_sum: %lu\n"
			"Join: %lu\n"
			"*************************************************\n"
#ifdef DECOMPOSED1_
			"Rebalance total: %lu\n"
#endif
			"Exp evaluation: %lu\n"
			"Remove empty total: %lu\n"
			"Data copy: %lu\n"
			"Total time: %lu\n", index_count_final, prefix_sum_final, join_final,
#ifdef DECOMPOSED1_
								rebalance_final,
#endif
								join_total, remove_empty_total, data_copy, total_);

}

extern "C" __global__ void EvaluateSearchPredicate(GTable outer_table, GExpressionVector search_keys, int outer_rows, int64_t *val_stack, ValueType *type_stack, GTable output, GHashIndex output_index)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

	GTuple tuple_res, outer_tuple;

	for (int i = index; i < outer_rows; i += stride) {
		tuple_res = output.getGTuple(i);
		outer_tuple = outer_table.getGTuple(i);

		for (int j = 0; j < search_keys.size(); j++) {
			GNValue eval_result = search_keys.at(j).evaluate(&outer_tuple, NULL, val_stack, type_stack, stride);

			tuple_res.setGNValue(eval_result, j);
		}

		output_index.insertKeyTupleNoSort(tuple_res, i);
	}
}

extern "C" __global__ void indexCount(GHashIndex outer_index, GHashIndex inner_index, ulong *index_count, ResBound *out_bound)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	int outer_rows = outer_index.getKeyRows();
	GHashIndexKey key;

	for (int i = index; i < outer_rows; i += stride) {
		key = outer_index.getKeyAtIndex(i);
		int bucket_id = key.KeyHasher();

		out_bound[i].left = inner_index.getBucketLocation(bucket_id);
		out_bound[i].right = inner_index.getBucketLocation(bucket_id + 1);

		index_count[i] = out_bound[i].right - out_bound[i].left + 1;
	}
}

void GPUHJ::IndexCount(ulong *index_count, ResBound *out_bound)
{
	int outer_rows = outer_table_.getCurrentRowNum();
	int block_x, grid_x;

	block_x = (outer_rows < BLOCK_SIZE_X) ? outer_rows : BLOCK_SIZE_X;
	grid_x = (outer_rows - 1)/block_x + 1;

	GColumnInfo *search_schema;

	checkCudaErrors(hipMalloc(&search_schema, sizeof(GColumnInfo) * search_exp_.size()));
	//GTable search_table(NULL, search_schema, search_exp_num_, outer_table_.getCurrentRowNum());
	GTable search_table(NULL, search_schema, search_exp_.size());
	GHashIndex tmp_index(outer_table_.getCurrentRowNum(), search_exp_.size(), maxNumberOfBuckets_);

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * outer_rows * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * outer_rows * MAX_STACK_SIZE));
	EvaluateSearchPredicate<<<grid_x, block_x>>>(outer_table_, search_exp_, outer_table_.getCurrentRowNum(), val_stack, type_stack, search_table, tmp_index);
	//GHashIndex *inner_index = dynamic_cast<GHashIndex*>(inner_table_.getCurrentIndex());
	GHashIndex *inner_index;
	indexCount<<<grid_x, block_x>>>(tmp_index, *inner_index, index_count, out_bound);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(search_schema));
	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

void GPUHJ::IndexCount(ulong *index_count, ResBound *out_bound, hipStream_t stream)
{
	int outer_rows = outer_table_.getCurrentRowNum();
	int block_x, grid_x;

	block_x = (outer_rows < BLOCK_SIZE_X) ? outer_rows : BLOCK_SIZE_X;
	grid_x = (outer_rows - 1)/block_x + 1;

	GTable search_table(NULL, search_exp_.size());

	GHashIndex tmp_index(outer_table_.getCurrentRowNum(), search_exp_.size(), maxNumberOfBuckets_);

	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * outer_rows * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * outer_rows * MAX_STACK_SIZE));

	EvaluateSearchPredicate<<<grid_x, block_x, 0, stream>>>(outer_table_, search_exp_, outer_table_.getCurrentRowNum(), val_stack, type_stack, search_table, tmp_index);
	//GHashIndex *inner_index = dynamic_cast<GHashIndex*>(inner_table_.getCurrentIndex());
	GHashIndex *inner_index;
	indexCount<<<grid_x, block_x, 0, stream>>>(tmp_index, *inner_index, index_count, out_bound);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	search_table.removeTable();
	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

extern "C" __global__ void hashJoinLegacy(GTable outer, GTable inner,
											RESULT *in_bound, RESULT *out_bound,
											ulong *mark_location, int size,
											GExpression end_exp, GExpression post_exp, GExpression where_exp,
											int64_t *val_stack, ValueType *type_stack
											)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int offset = blockDim.x * gridDim.x;
	GNValue res;
	GTuple outer_tuple, inner_tuple;

	for (int i = index; i < size; i += offset) {
		outer_tuple = outer.getGTuple(in_bound[i].lkey);
		inner_tuple = inner.getGTuple(in_bound[i].rkey);
		res = GNValue::getTrue();

		res = (end_exp.getSize() > 0) ? end_exp.evaluate(&outer_tuple, &inner_tuple, val_stack + index, type_stack + index, offset) : res;
		res = (post_exp.getSize() > 0 && res.isTrue()) ? post_exp.evaluate(&outer_tuple, &inner_tuple, val_stack + index, type_stack + index, offset) : res;
		res = (where_exp.getSize() > 0 && res.isTrue()) ? where_exp.evaluate(&outer_tuple, &inner_tuple, val_stack + index, type_stack + index, offset) : res;

		out_bound[i].lkey = (res.isTrue()) ? in_bound[i].lkey : (-1);
		out_bound[i].rkey = (res.isTrue()) ? in_bound[i].rkey : (-1);
		mark_location[i] = (res.isTrue()) ? 1 : 0;
	}

	if (index == 0) {
		mark_location[size] = 0;
	}
}

void GPUHJ::HashJoinLegacy(RESULT *in_bound, RESULT *out_bound, ulong *mark_location, int size)
{
	int partition_size = DEFAULT_PART_SIZE_;
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size < partition_size) ? (size - 1)/block_x + 1 : (partition_size - 1)/block_x + 1;


	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 block_size(block_x, 1, 1);
	dim3 grid_size(grid_x, 1, 1);

	hashJoinLegacy<<<grid_size, block_size>>>(outer_table_, inner_table_,
												in_bound, out_bound,
												mark_location, size,
												end_expression_, post_expression_, where_expression_,
												val_stack,
												type_stack);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}

void GPUHJ::HashJoinLegacy(RESULT *in_bound, RESULT *out_bound, ulong *mark_location, int size, hipStream_t stream)
{
	int partition_size = DEFAULT_PART_SIZE_;
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size < partition_size) ? (size - 1)/block_x + 1 : (partition_size - 1)/block_x + 1;


	int64_t *val_stack;
	ValueType *type_stack;

	checkCudaErrors(hipMalloc(&val_stack, sizeof(int64_t) * block_x * grid_x * MAX_STACK_SIZE));
	checkCudaErrors(hipMalloc(&type_stack, sizeof(ValueType) * block_x * grid_x * MAX_STACK_SIZE));

	dim3 block_size(block_x, 1, 1);
	dim3 grid_size(grid_x, 1, 1);

	hashJoinLegacy<<<grid_size, block_size, 0, stream>>>(outer_table_, inner_table_,
															in_bound, out_bound,
															mark_location, size,
															end_expression_, post_expression_, where_expression_,
															val_stack,
															type_stack);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipFree(val_stack));
	checkCudaErrors(hipFree(type_stack));
}


__global__ void HashDecompose(RESULT *output, ResBound *in_bound, int *sorted_idx, ulong *in_location, ulong *local_offset, int size)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = index; i < size; i += blockDim.x * gridDim.x) {
		output[i].lkey = in_bound[in_location[i]].outer;
		output[i].rkey = sorted_idx[in_bound[in_location[i]].left + local_offset[i]];
	}
}

void GPUHJ::decompose(RESULT *output, ResBound *in_bound, ulong *in_location, ulong *local_offset, int size)
{
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size - 1)/block_x + 1;

	dim3 block_size(block_x, 1, 1);
	dim3 grid_size(grid_x, 1, 1);

	//GHashIndex *inner_idx = dynamic_cast<GHashIndex *>(inner_table_.getCurrentIndex());
	GHashIndex *inner_idx;
	int *sorted_idx = inner_idx->getSortedIdx();

	HashDecompose<<<grid_size, block_size>>>(output, in_bound, sorted_idx, in_location, local_offset, size);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());
}

void GPUHJ::decompose(RESULT *output, ResBound *in_bound, ulong *in_location, ulong *local_offset, int size, hipStream_t stream)
{
	int block_x, grid_x;

	block_x = (size < BLOCK_SIZE_X) ? size : BLOCK_SIZE_X;
	grid_x = (size - 1)/block_x + 1;

	dim3 block_size(block_x, 1, 1);
	dim3 grid_size(grid_x, 1, 1);

	//GHashIndex *inner_idx = dynamic_cast<GHashIndex *>(inner_table_.getCurrentIndex());
	GHashIndex *inner_idx;
	int *sorted_idx = inner_idx->getSortedIdx();

	HashDecompose<<<grid_size, block_size, 0, stream>>>(output, in_bound, sorted_idx, in_location, local_offset, size);
	checkCudaErrors(hipGetLastError());
}

void GPUHJ::Rebalance(ulong *index_count, ResBound *in_bound, RESULT **out_bound, int in_size, ulong *out_size)
{
	GUtilities::ExclusiveScan(index_count, in_size, out_size);

	if (*out_size == 0) {
		return;
	}

	ulong *location;

	checkCudaErrors(hipMalloc(&location, sizeof(ulong) * (*out_size)));
	checkCudaErrors(hipMemset(location, 0, sizeof(ulong) * (*out_size)));
	checkCudaErrors(hipDeviceSynchronize());


	GUtilities::MarkLocation(location, index_count, in_size);


	GUtilities::InclusiveScan(location, *out_size);

	ulong *local_offset;

	checkCudaErrors(hipMalloc(&local_offset, *out_size * sizeof(ulong)));
	checkCudaErrors(hipMalloc(out_bound, *out_size * sizeof(RESULT)));

	GUtilities::ComputeOffset(index_count, location, local_offset, *out_size);

	decompose(*out_bound, in_bound, location, local_offset, *out_size);

	checkCudaErrors(hipFree(local_offset));
	checkCudaErrors(hipFree(location));
}

void GPUHJ::Rebalance(ulong *index_count, ResBound *in_bound, RESULT **out_bound, int in_size, ulong *out_size, hipStream_t stream)
{
	GUtilities::ExclusiveScan(index_count, in_size, out_size, stream);

	if (*out_size == 0) {
		return;
	}

	ulong *location;

	checkCudaErrors(hipMalloc(&location, sizeof(ulong) * (*out_size)));
	checkCudaErrors(hipMemsetAsync(location, 0, sizeof(ulong) * (*out_size), stream));

	GUtilities::MarkLocation(location, index_count, in_size, stream);

	GUtilities::InclusiveScan(location, *out_size, stream);

	ulong *local_offset;

	checkCudaErrors(hipMalloc(&local_offset, *out_size * sizeof(ulong)));
	checkCudaErrors(hipMalloc(out_bound, *out_size * sizeof(RESULT)));

	GUtilities::ComputeOffset(index_count, location, local_offset, *out_size, stream);

	decompose(*out_bound, in_bound, location, local_offset, *out_size, stream);

	checkCudaErrors(hipFree(local_offset));
	checkCudaErrors(hipFree(location));
}

}
